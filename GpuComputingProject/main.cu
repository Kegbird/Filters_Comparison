﻿#include <stdio.h>
#include<math.h>
#include <time.h>
#include "utils.h"
#include "cpu_imp.cuh"
#include "gpu_imp.cuh"

#define KERNEL_SIZE 3
#define KERNEL_RADIUS KERNEL_SIZE/2

#define GAUSSIAN_KERNEL_SIZE 7
#define GAUSSIAN_KERNEL_RADIUS GAUSSIAN_KERNEL_SIZE/2
#define SIGMA 1
#define LOW_THRESHOLD_RATIO 0.05
#define HIGH_THRESHOLD_RATIO 0.5
#define OUTPUT true

float sobel_kernel_3x3_h[3][3] = { {1, 0, -1}, {2, 0, -2}, {1, 0, -1} };
float sobel_kernel_3x3_v[3][3] = { {1, 2, 1}, {0, 0, 0}, {-1, -2, -1} };

float robert_kernel_3x3_h[3][3] = { {1, 0, 0}, {0, -1, 0}, {0, 0, 0} };
float robert_kernel_3x3_v[3][3] = { {0, 1, 0},{-1, 0, 0}, {0, 0, 0} };

float gaussian_kernel_7x7[7][7];

char filename[] = "A.jpg";

void print_device_props()
{
	printf("============================\n");
	printf("	Device info	\n");
	printf("============================\n\n");
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	printf("Device name: %s\n", prop.name);
	printf("Memory Clock Rate (KHz): %d\n",
		prop.memoryClockRate);
	printf("Memory Bus Width (bits): %d\n",
		prop.memoryBusWidth);
	printf("Peak Memory Bandwidth (GB/s): %f\n\n",
		2.0*prop.memoryClockRate*(prop.memoryBusWidth / 8) / 1.0e6);
}

int main()
{
	print_device_props();

	if (!check_input(filename))
		return 0;
	print_file_details(filename);

	printf("============================\n");
	printf("	CPU Convolution(Robert)	\n");
	printf("============================\n\n");
	filter_cpu(filename, "Sample_Robert.png", &robert_kernel_3x3_h[0][0], KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);

	load_constant_memory_robert_h(&robert_kernel_3x3_h[0][0], KERNEL_SIZE);
	load_constant_memory_robert_v(&robert_kernel_3x3_v[0][0], KERNEL_SIZE);

	load_constant_memory_sobel_h(&sobel_kernel_3x3_h[0][0], KERNEL_SIZE);
	load_constant_memory_sobel_v(&sobel_kernel_3x3_v[0][0], KERNEL_SIZE);

	/*printf("============================\n");
	printf("	GPU Convolution(Robert) - Parallel	\n");
	printf("============================\n\n");

	naive_robert_convolution_gpu(filename, KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);

	printf("============================\n");
	printf("	GPU Convolution(Robert) - Smem	\n");
	printf("============================\n\n");

	smem_robert_convolution_gpu(filename, KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);

	printf("============================\n");
	printf("	GPU Convolution(Robert) - Streams	\n");
	printf("============================\n\n");

	stream_robert_convolution_gpu(filename, KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);

	printf("============================\n");
	printf("	CPU Module(Sobel)\n");
	printf("============================\n\n");

	module_cpu(filename, "Sample_Module_Sobel.png", &sobel_kernel_3x3_h[0][0], &sobel_kernel_3x3_v[0][0], KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);

	printf("============================\n");
	printf("	GPU Module(Sobel) - Parallel \n");
	printf("============================\n\n");

	naive_sobel_module_gpu(filename, KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);

	printf("============================\n");
	printf("	GPU Module(Sobel) - Smem	\n");
	printf("============================\n\n");

	smem_sobel_module_gpu(filename, KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);

	printf("============================\n");
	printf("	GPU Module(Sobel) - Streams	\n");
	printf("============================\n\n");

	stream_sobel_module_gpu(filename, KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);
	*/
	calculate_gaussian_kernel(&gaussian_kernel_7x7[0][0], SIGMA, GAUSSIAN_KERNEL_SIZE, GAUSSIAN_KERNEL_RADIUS);
	load_constant_memory_gaussian(&gaussian_kernel_7x7[0][0], GAUSSIAN_KERNEL_SIZE);
	
	printf("============================\n");
	printf("	CPU Canny Filter \n");
	printf("============================\n\n");

	canny_cpu(filename, "Sample_Canny.png", &sobel_kernel_3x3_h[0][0], &sobel_kernel_3x3_v[0][0], &gaussian_kernel_7x7[0][0], SIGMA, GAUSSIAN_KERNEL_SIZE, GAUSSIAN_KERNEL_RADIUS, LOW_THRESHOLD_RATIO, HIGH_THRESHOLD_RATIO, OUTPUT);

	printf("============================\n");
	printf("	GPU Canny Filter - Parallel	\n");
	printf("============================\n\n");

	naive_canny_gpu(filename, SIGMA, GAUSSIAN_KERNEL_SIZE, GAUSSIAN_KERNEL_RADIUS, LOW_THRESHOLD_RATIO, HIGH_THRESHOLD_RATIO, OUTPUT);

	printf("============================\n");
	printf("	GPU Canny Filter - Smem	\n");
	printf("============================\n\n");

	smem_canny_gpu(filename, SIGMA, GAUSSIAN_KERNEL_SIZE, GAUSSIAN_KERNEL_RADIUS, LOW_THRESHOLD_RATIO, HIGH_THRESHOLD_RATIO, OUTPUT);

	return 0;
}

