#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<math.h>
#define STB_IMAGE_IMPLEMENTATION 
#include <stb_image.h>
#define STB_IMAGE_WRITE_IMPLEMENTATION 
#include <stb_image_write.h>
#include <time.h>

#define BLOCK_SIZE 32
#define KERNEL_SIZE 3
#define KERNEL_RADIUS KERNEL_SIZE/2

int robert_kernel_3x3_h[3][3] = { {1, 0, 0}, {0, -1, 0}, {0, 0, 0} };
int robert_kernel_3x3_v[3][3] = { {0, 1, 0},{-1, 0, 0}, {0, 0, 0} };

__constant__ int d_robert_kernel_3x3_h[3][3];
__constant__ int d_robert_kernel_3x3_v[3][3];

int sobel_kernel_3x3_h[3][3] = { {1, 0, -1}, {2, 0, -2}, {1, 0, -1} };
int sobel_kernel_3x3_v[3][3] = { {1, 2, 1}, {0, 0, 0}, {-1, -2, -1} };

unsigned char* image;
unsigned char* d_image;
unsigned char* filtered_image;
unsigned char* d_filtered_image;

#pragma region CPU_IMPLEMENTATIONS

int cpu_convolution(unsigned char* pixel, int channels, int* kernel, int width, int height, int kernel_size)
{
	int result = 0;
	int kernel_length = kernel_size * kernel_size;
	int color = 0;
	for (int j = 0; j < kernel_size; j++)
	{
		for (int k = 0; k < kernel_size; k++)
		{
			for (int i = 0; i < channels; i++)
				color += pixel[i];
			color /= channels;
			result += color * kernel[j*kernel_size + k];
			pixel += channels;
			color = 0;
		}
		pixel += (width * channels) - channels * (kernel_size - 1);
	}

	return result;
}


void cpu_filter(unsigned char* image, int width, int height, int channels, size_t image_size, int* kernel, int kernel_size, unsigned char* result)
{
	unsigned char* pixel = image;
	unsigned char* r = result;
	int kernel_radius = kernel_size / 2;
	int value = 0;
	for (int i = 0; i < height - kernel_radius * 2; i++)
	{
		for (int j = 0; j < width - kernel_radius * 2; j++, pixel += channels)
		{
			value = cpu_convolution(pixel, channels, kernel, width, height, kernel_size);
			if (value < 0)
				r[0] = 0;
			else
				r[0] = value;
			r += 1;
		}
		pixel += (kernel_radius * channels) * 2;
	}
}

void cpu_module(unsigned char* image, int width, int height, int channels, size_t image_size, int* kernel_h, int* kernel_v, int kernel_size, unsigned char* result)
{
	int gh = 0;
	int gv = 0;
	int modulo = 0;
	int kernel_radius = kernel_size / 2;

	unsigned char* pixel = image;
	unsigned char* r = result;
	for (int i = 0; i < height - kernel_radius * 2; i++)
	{
		for (int j = 0; j < width - kernel_radius * 2; j++, pixel += channels)
		{
			gh = cpu_convolution(pixel, channels, kernel_h, width, height, kernel_size);
			gv = cpu_convolution(pixel, channels, kernel_v, width, height, kernel_size);
			r[0] = sqrt(gh*gh + gv * gv);
			r += 1;
		}
		pixel += (kernel_radius * channels) * 2;
	}
}

#pragma endregion

#pragma region GPU_IMPLEMENTATIONS
__global__ void kernel_robert_h_convolution(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - (KERNEL_RADIUS) * 2 <= col || height <= row - (KERNEL_RADIUS) * 2)
		return;

	int color = 0;
	int result = 0;
	int index = row * width + col - ((KERNEL_RADIUS) * 2)*row;
	unsigned char* pixel = image + row * width * channels + col * channels;
	for (int i = 0; i < KERNEL_SIZE; i++)
	{
		for (int j = 0; j < KERNEL_SIZE; j++)
		{
			for (int k = 0; k < channels; k++)
				color += pixel[k];
			color /= 3;
			result += color * d_robert_kernel_3x3_h[i][j];
			pixel += channels;
			color = 0;
		}
		pixel += (width * channels) - channels * (KERNEL_SIZE - 1);
	}
	if (result < 0)
		result = 0;
	(filtered_image + index)[0] = result;
}

#pragma endregion

void freeHostMemory()
{
	free(image);
	free(filtered_image);
}

void freeDeviceMemory()
{
	hipFree(d_image);
	hipFree(d_filtered_image);
}

int main()
{
	dim3 grid;
	dim3 block;
	int width;
	int height;
	int f_width;
	int f_height;
	int channels;
	size_t image_size;
	size_t filtered_image_size;
	char filename[] = "Sample.png";
	time_t begin;
	time_t end;
	double cpu_time = 1;
	double elapsed_time = 0;
	hipError_t status;
	//Image loading and common check
	image = stbi_load(filename, &width, &height, &channels, 0);
	if (image == NULL)
	{
		printf("No image provided!");
		return 0;
	}
	if (width < 3 || height < 3)
	{
		printf("The image provided is too small; the minimum resolution is 3x3 pixels.\n");
		return 0;
	}
	image_size = width * height * channels;
	printf("============================\n");
	printf("	Input Details	\n");
	printf("============================\n\n");
	printf("Width: %d\n", width);
	printf("Height: %d\n", height);
	printf("Channels: %d\n", channels);
	printf("Size: %d bytes\n\n", image_size);
	//Convolution decreases the resolution of the result
	f_width = width - (KERNEL_SIZE / 2) * 2;
	f_height = height - (KERNEL_SIZE / 2) * 2;
	filtered_image_size = f_width * f_height;
	filtered_image = (unsigned char*)malloc(filtered_image_size);
	//initialization of constant memory
	hipMemcpyToSymbol(HIP_SYMBOL(d_robert_kernel_3x3_h), &robert_kernel_3x3_h, 3 * 3 * sizeof(int));
	hipMemcpyToSymbol(HIP_SYMBOL(d_robert_kernel_3x3_v), &robert_kernel_3x3_v, 3 * 3 * sizeof(int));
	printf("============================\n");
	printf("	CPU Convolution(Robert)	\n");
	printf("============================\n\n");
	begin = clock();
	cpu_filter(image, width, height, channels, image_size, &(robert_kernel_3x3_h[0][0]), KERNEL_SIZE, filtered_image);
	end = clock();
	cpu_time = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("CPU Convolution:%f seconds\n\n", cpu_time);
	stbi_write_png("Sample_Convolution_Robert.png", f_width, f_height, 1, filtered_image, f_width);

	printf("============================\n");
	printf("	GPU naive Convolution(Robert)	\n");
	printf("============================\n\n");
	printf("Allocation of image on GPU gmem...\n\n");
	//Allocation gmem
	status = hipMalloc((void**)&d_image, image_size);
	if (status != hipSuccess)
	{
		printf("First hipMalloc failed!\n");
		return 0;
	}

	status = hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice);
	if (status != hipSuccess)
	{
		printf("First hipMemcpy failed!\n");
		return 0;
	}

	status = hipMalloc((void**)&d_filtered_image, filtered_image_size);
	if (status != hipSuccess)
	{
		printf("Second hipMalloc failed!\n");
		return 0;
	}

	status=hipMemset(d_filtered_image, 0, filtered_image_size);

	if (status != hipSuccess)
	{
		printf("First hipMemset failed!\n");
		return 0;
	}

	printf("32x32 blocks\n");
	block = dim3(BLOCK_SIZE, BLOCK_SIZE);
	grid = dim3((f_width + block.x - 1) / block.x, (f_height + block.y - 1) / block.y);
	begin = clock();
	kernel_robert_h_convolution << < grid, block >> > (d_image, d_filtered_image, width, height, channels);
	hipDeviceSynchronize();
	end = clock();
	status = hipMemcpy(filtered_image, d_filtered_image, filtered_image_size, hipMemcpyDeviceToHost);

	/*if (status != hipSuccess)
	{
		printf(hipGetErrorString(status));
		return 0;
	}*/

	stbi_write_png("Sample_Naive_Convolution_Robert_32x32_block.png", f_width, f_height, 1, filtered_image, f_width);
	elapsed_time = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("GPU naive Convolution:%f seconds\n", elapsed_time);
	printf("Speedup: %f\n\n", elapsed_time / cpu_time);

	printf("============================\n");
	printf("	CPU Module(Sobel)	\n");
	printf("============================\n\n");
	begin = clock();
	cpu_module(image, width, height, channels, image_size, &(sobel_kernel_3x3_h[0][0]), &(sobel_kernel_3x3_v[0][0]), KERNEL_SIZE, filtered_image);
	end = clock();
	elapsed_time = (double)(end - begin) / CLOCKS_PER_SEC;
	printf("CPU Module:%f seconds\n\n", elapsed_time);
	stbi_write_png("Sample_Module_Sobel.png", f_width, f_height, 1, filtered_image, f_width);

	freeHostMemory();
	freeDeviceMemory();
	return 0;
}

