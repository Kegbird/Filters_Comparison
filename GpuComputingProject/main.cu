﻿#include <stdio.h>
#include<math.h>
#include <time.h>
#include "utils.h"
#include "cpu_imp.cuh"
#include "gpu_imp.cuh"

#define KERNEL_SIZE 3
#define KERNEL_RADIUS KERNEL_SIZE/2

#define GAUSSIAN_KERNEL_SIZE 7
#define GAUSSIAN_KERNEL_RADIUS GAUSSIAN_KERNEL_SIZE/2
#define SIGMA 1
#define LOW_THRESHOLD_RATIO 0.05
#define HIGH_THRESHOLD_RATIO 0.5
#define OUTPUT true

float sobel_kernel_3x3_h[3][3] = { {1, 0, -1}, {2, 0, -2}, {1, 0, -1} };
float sobel_kernel_3x3_v[3][3] = { {1, 2, 1}, {0, 0, 0}, {-1, -2, -1} };

float robert_kernel_3x3_h[3][3] = { {1, 0, 0}, {0, -1, 0}, {0, 0, 0} };
float robert_kernel_3x3_v[3][3] = { {0, 1, 0},{-1, 0, 0}, {0, 0, 0} };

float gaussian_kernel_7x7[7][7];


char filename[] = "Prova.png";

int main()
{
	if (!check_input(filename))
		return 0;
	print_file_details(filename);

	printf("============================\n");
	printf("	CPU Convolution(Robert)	\n");
	printf("============================\n\n");
	filter_cpu(filename, "Sample_Robert.png", &robert_kernel_3x3_h[0][0], KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);

	load_constant_memory_robert_h(&robert_kernel_3x3_h[0][0], KERNEL_SIZE);
	load_constant_memory_robert_v(&robert_kernel_3x3_v[0][0], KERNEL_SIZE);

	load_constant_memory_sobel_h(&sobel_kernel_3x3_h[0][0], KERNEL_SIZE);
	load_constant_memory_sobel_v(&sobel_kernel_3x3_v[0][0], KERNEL_SIZE);

	/*printf("============================\n");
	printf("	GPU Convolution(Robert) - Parallel	\n");
	printf("============================\n\n");

	naive_robert_convolution_gpu(filename, KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);

	printf("============================\n");
	printf("	GPU Convolution(Robert) - Smem	\n");
	printf("============================\n\n");

	smem_robert_convolution_gpu(filename, KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);

	printf("============================\n");
	printf("	GPU Convolution(Robert) - Streams	\n");
	printf("============================\n\n");

	stream_robert_convolution_gpu(filename, KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);

	printf("============================\n");
	printf("	CPU Module(Sobel)\n");
	printf("============================\n\n");

	module_cpu(filename, "Sample_Module_Sobel.png", &sobel_kernel_3x3_h[0][0], &sobel_kernel_3x3_v[0][0], KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);

	printf("============================\n");
	printf("	GPU Module(Sobel) - Parallel \n");
	printf("============================\n\n");

	naive_sobel_module_gpu(filename, KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);

	printf("============================\n");
	printf("	GPU Module(Sobel) - Smem	\n");
	printf("============================\n\n");

	smem_sobel_module_gpu(filename, KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);

	printf("============================\n");
	printf("	GPU Module(Sobel) - Streams	\n");
	printf("============================\n\n");

	stream_sobel_module_gpu(filename, KERNEL_SIZE, KERNEL_RADIUS, OUTPUT);*/

	calculate_gaussian_kernel(&gaussian_kernel_7x7[0][0], SIGMA, GAUSSIAN_KERNEL_SIZE, GAUSSIAN_KERNEL_RADIUS);
	load_constant_memory_gaussian(&gaussian_kernel_7x7[0][0], GAUSSIAN_KERNEL_SIZE);

	printf("============================\n");
	printf("	CPU Canny Filter \n");
	printf("============================\n\n");

	canny_cpu(filename, "Sample_Canny.png", &sobel_kernel_3x3_h[0][0], &sobel_kernel_3x3_v[0][0], &gaussian_kernel_7x7[0][0], SIGMA, GAUSSIAN_KERNEL_SIZE, GAUSSIAN_KERNEL_RADIUS, LOW_THRESHOLD_RATIO, HIGH_THRESHOLD_RATIO, OUTPUT);

	printf("============================\n");
	printf("	GPU Canny Filter - Parallel	\n");
	printf("============================\n\n");
	naive_canny_gpu(filename, &sobel_kernel_3x3_h[0][0], &sobel_kernel_3x3_v[0][0], &gaussian_kernel_7x7[0][0], SIGMA, GAUSSIAN_KERNEL_SIZE, GAUSSIAN_KERNEL_RADIUS, LOW_THRESHOLD_RATIO, HIGH_THRESHOLD_RATIO, OUTPUT);

	return 0;
}

