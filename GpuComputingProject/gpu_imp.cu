#include "hip/hip_runtime.h"
﻿#include"gpu_imp.cuh"
#include "utils.h"
#include <stdio.h>
#include <time.h>
#include "math.h"
#include "hip/hip_runtime.h"
#include ""

#define STREAMS 2
#define BLOCK_SIZES 2
__constant__ float d_robert_kernel_3x3_h[3][3];
__constant__ float d_robert_kernel_3x3_v[3][3];

__constant__ float d_sobel_kernel_3x3_h[3][3];
__constant__ float d_sobel_kernel_3x3_v[3][3];

__constant__ float d_gaussian_kernel_7x7[7][7];

const int block_sizes[BLOCK_SIZES] = { 16, 32 };

unsigned char* image;
unsigned char* filtered_image;
unsigned char* pinned_image;
unsigned char* pinned_filtered_image;
unsigned char* d_image;
unsigned char* d_filtered_image;
unsigned char* d_gaussian_image;
unsigned char* d_module_image;
unsigned char* d_non_max_image;

float* d_orientations;
int width;
int height;
int f_width;
int f_height;
int channels;
size_t image_size;
size_t filtered_image_size;

const char* output_filename_robert[] = { "Sample_Naive_Convolution_Robert_16x16_block.png",
									"Sample_Naive_Convolution_Robert_32x32_block.png" };

const char* output_filename_robert_smem[] = { "Sample_Convolution_Robert_Smem_16x16_block.png",
									"Sample_Convolution_Robert_Smem_32x32_block.png" };

const char* output_filename_robert_stream[] = { "Sample_Convolution_Robert_Stream_16x16_block.png",
									"Sample_Convolution_Robert_Stream_32x32_block.png" };


const char* output_filename_module[] = { "Sample_Naive_Sobel_Module_16x16_block.png",
									"Sample_Naive_Sobel_Module_32x32_block.png" };

const char* output_filename_module_smem[] = { "Sample_Sobel_Module_Smem_16x16_block.png",
									"Sample_Sobel_Module_Smem_32x32_block.png" };

const char* output_filename_module_stream[] = { "Sample_Sobel_Module_Stream_16x16_block.png",
									"Sample_Sobel_Module_Stream_32x32_block.png" };

const char* output_filename_canny[] = { "Sample_Canny_16x16_block.png",
										"Sample_Canny_32x32_block.png" };

__device__ float grayscale(unsigned char* pixel, int channels)
{
	float color = 0;
	for (int j = 0; j < channels; j++)
		color += pixel[j] / channels;
	return color;
}

__device__ float convolution(unsigned char* pixel, int channels, int width, float* kernel, int kernel_size, int kernel_radius)
{
	float result = 0;
	for (int i = 0; i < kernel_size; i++)
	{
		for (int j = 0; j < kernel_size; j++)
		{
			result += grayscale(pixel, channels) * kernel[i*kernel_size + j];
			pixel += channels;
		}
		pixel += (width * channels) - channels * (kernel_size - 1) - channels;
	}
	if (result < 0)
		result = 0;
	return result;
}

__device__ bool strong_neighbour(unsigned char* pixel, int width, int strong_color)
{
	if (*(pixel - width - 1) == strong_color || *(pixel - width) == strong_color || *(pixel - width + 1) == strong_color ||
		*(pixel - 1) == strong_color || *(pixel + 1) == strong_color ||
		*(pixel + width - 1) == strong_color || *(pixel + width) == strong_color || *(pixel + width + 1) == strong_color)
		return true;
	return false;
}

__device__ int module(unsigned char* pixel, int channels, int width, float* kernel_h, float* kernel_v, int kernel_size, int kernel_radius)
{
	float gh = 0.0, gv = 0.0;
	for (int i = 0; i < kernel_size; i++)
	{
		//Evaluating gh and gv
		for (int j = 0; j < kernel_size; j++)
		{
			gh += grayscale(pixel, channels) * kernel_h[i*kernel_size + j];
			gv += grayscale(pixel, channels) * kernel_v[i*kernel_size + j];
			pixel += channels;
		}
		pixel += (width * channels) - channels * (kernel_size - 1) - channels;
	}

	return sqrtf(gh*gh + gv * gv);
}

__global__ void kernel_robert_h_convolution(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels, int kernel_size, int kernel_radius)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - (kernel_radius) * 2 <= col || height - (kernel_radius) * 2 <= row)
		return;

	int index = row * width + col - ((kernel_radius) * 2)*row;
	unsigned char* pixel = image + row * width * channels + col * channels;
	(filtered_image + index)[0] = convolution(pixel, channels, width, &d_robert_kernel_3x3_h[0][0], kernel_size, kernel_radius);
}

__global__ void kernel_robert_h_convolution_smem(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels, int tile_side, int kernel_size, int kernel_radius)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if ((width - (kernel_radius) * 2 <= col || height - (kernel_radius) * 2 <= row))
		return;

	extern __shared__ unsigned char image_tile[];

	unsigned char *pixel = image + row * width *channels + col * channels;

	int tile_index = threadIdx.y*tile_side + threadIdx.x;

	image_tile[tile_index] = grayscale(pixel, channels);

	if ((threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1) || (row == height - (kernel_radius) * 2 - 1) || (col == width - (kernel_radius) * 2 - 1))
	{
		//Bottom right corner thread
		image_tile[tile_index + 1] = grayscale(pixel + channels, channels);
		image_tile[tile_index + 2] = grayscale(pixel + channels * 2, channels);
		image_tile[tile_index + tile_side] = grayscale(pixel + width * channels, channels);
		image_tile[tile_index + tile_side * 2] = grayscale(pixel + (width*channels) * 2, channels);

		image_tile[tile_index + tile_side + 1] = grayscale(pixel + width * channels + channels, channels);
		image_tile[tile_index + tile_side + 2] = grayscale(pixel + width * channels + channels * 2, channels);
		image_tile[tile_index + tile_side * 2 + 1] = grayscale(pixel + width * channels * 2 + channels, channels);
		image_tile[tile_index + tile_side * 2 + 2] = grayscale(pixel + width * channels * 2 + channels * 2, channels);
	}
	else if (threadIdx.x == blockDim.x - 1 || (col == width - (kernel_radius) * 2 - 1))
	{
		//Right edge thread
		image_tile[tile_index + 1] = grayscale(pixel + channels, channels);
		image_tile[tile_index + 2] = grayscale(pixel + channels * 2, channels);
	}
	else if (threadIdx.y == blockDim.y - 1 || (row == height - (kernel_radius) * 2 - 1))
	{
		//Bottom left corner thread
		image_tile[tile_index + tile_side] = grayscale(pixel + width * channels, channels);
		image_tile[tile_index + tile_side * 2] = grayscale(pixel + (width*channels) * 2, channels);
	}

	__syncthreads();

	int result = 0;

	for (int i = 0; i < kernel_size; i++)
	{
		for (int j = 0; j < kernel_size; j++, tile_index++)
			result += image_tile[tile_index] * d_robert_kernel_3x3_h[i][j];
		tile_index += tile_side - kernel_radius * 2 - 1;
	}
	if (result < 0)
		result = 0;

	int index = row * width + col - ((kernel_radius) * 2)*row;
	(filtered_image + index)[0] = result;
}

__global__ void kernel_robert_h_convolution_stream(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels, int offset_input, int offset_output, int kernel_size, int kernel_radius)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - (kernel_radius) * 2 <= col || height - (kernel_radius) * 2 < row)
		return;

	unsigned char* pixel = image + row * width * channels + col * channels + offset_input;
	int index = offset_output + (row * width + col - ((kernel_radius) * 2)*row);
	(filtered_image + index)[0] = convolution(pixel, channels, width, &d_robert_kernel_3x3_h[0][0], kernel_size, kernel_radius);
}

__global__ void kernel_module_sobel(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels, int kernel_size, int kernel_radius)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - (kernel_radius) * 2 <= col || height - (kernel_radius) * 2 <= row)
		return;

	int index = row * width + col - ((kernel_radius) * 2)*row;
	unsigned char* pixel = image + row * width * channels + col * channels;
	(filtered_image + index)[0] = module(pixel, channels, width, &d_sobel_kernel_3x3_h[0][0], &d_sobel_kernel_3x3_v[0][0], kernel_size, kernel_radius);
}

__global__ void kernel_module_sobel_smem(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels, int tile_side, int kernel_size, int kernel_radius)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if ((width - (kernel_radius) * 2 <= col || height - (kernel_radius) * 2 <= row))
		return;

	extern __shared__ unsigned char image_tile[];

	unsigned char *pixel = image + row * width *channels + col * channels;

	int tile_index = threadIdx.y*tile_side + threadIdx.x;
	//Image_tile contains the grayscale portion of the image on which the module will be applied
	image_tile[tile_index] = grayscale(pixel, channels);

	if ((threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1) || (row == height - (kernel_radius) * 2 - 1) || (col == width - (kernel_radius) * 2 - 1))
	{
		//Bottom right corner thread
		image_tile[tile_index + 1] = grayscale(pixel + channels, channels);
		image_tile[tile_index + 2] = grayscale(pixel + channels * 2, channels);
		image_tile[tile_index + tile_side] = grayscale(pixel + width * channels, channels);
		image_tile[tile_index + tile_side * 2] = grayscale(pixel + (width*channels) * 2, channels);

		image_tile[tile_index + tile_side + 1] = grayscale(pixel + width * channels + channels, channels);
		image_tile[tile_index + tile_side + 2] = grayscale(pixel + width * channels + channels * 2, channels);
		image_tile[tile_index + tile_side * 2 + 1] = grayscale(pixel + width * channels * 2 + channels, channels);
		image_tile[tile_index + tile_side * 2 + 2] = grayscale(pixel + width * channels * 2 + channels * 2, channels);
	}
	else if (threadIdx.x == blockDim.x - 1 || (col == width - (kernel_radius) * 2 - 1))
	{
		//Right edge thread
		image_tile[tile_index + 1] = grayscale(pixel + channels, channels);
		image_tile[tile_index + 2] = grayscale(pixel + channels * 2, channels);
	}
	else if (threadIdx.y == blockDim.y - 1 || (row == height - (kernel_radius) * 2 - 1))
	{
		//Bottom left corner thread
		image_tile[tile_index + tile_side] = grayscale(pixel + width * channels, channels);
		image_tile[tile_index + tile_side * 2] = grayscale(pixel + (width*channels) * 2, channels);
	}

	__syncthreads();

	int gh = 0;
	int gv = 0;
	for (int i = 0; i < kernel_size; i++)
	{
		for (int j = 0; j < kernel_size; j++, tile_index++)
		{
			gh += image_tile[tile_index] * d_sobel_kernel_3x3_h[i][j];
			gv += image_tile[tile_index] * d_sobel_kernel_3x3_v[i][j];
		}
		tile_index += tile_side - kernel_radius * 2 - 1;
	}

	int result = sqrtf(gh*gh + gv * gv);
	int index = row * width + col - ((kernel_radius) * 2)*row;
	(filtered_image + index)[0] = result;
}

__global__ void kernel_module_sobel_stream(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels, int offset_input, int offset_output, int kernel_size, int kernel_radius)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - (kernel_radius) * 2 <= col || height - (kernel_radius) * 2 < row)
		return;

	unsigned char* pixel = image + row * width * channels + col * channels + offset_input;
	int index = offset_output + (row * width + col - ((kernel_radius) * 2)*row);
	(filtered_image + index)[0] = module(pixel, channels, width, &d_sobel_kernel_3x3_h[0][0], &d_sobel_kernel_3x3_v[0][0], kernel_size, kernel_radius);
}

__global__ void kernel_gaussian_convolution(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels, int kernel_size, int kernel_radius)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - (kernel_radius) * 2 <= col || height - (kernel_radius) * 2 <= row)
		return;

	int index = row * width + col - ((kernel_radius) * 2)*row;
	unsigned char* pixel = image + row * width * channels + col * channels;
	(filtered_image + index)[0] = convolution(pixel, channels, width, &d_gaussian_kernel_7x7[0][0], kernel_size, kernel_radius);
}

__global__ void kernel_module_orientation(unsigned char* gaussian_filtered_image, unsigned char* module_image, float* orientations, int width, int height, int channels, int kernel_size, int kernel_radius)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - (kernel_radius) * 2 <= col || height - (kernel_radius) * 2 <= row)
		return;

	int index = row * width + col - ((kernel_radius) * 2)*row;
	unsigned char* pixel = gaussian_filtered_image + row * width * channels + col * channels;

	float gh = 0.0, gv = 0.0;
	for (int i = 0; i < kernel_size; i++)
	{
		//Evaluating gh and gv
		for (int j = 0; j < kernel_size; j++)
		{
			gh += grayscale(pixel, channels) * d_sobel_kernel_3x3_h[i][j];
			gv += grayscale(pixel, channels) * d_sobel_kernel_3x3_v[i][j];
			pixel += channels;
		}
		pixel += (width * channels) - channels * (kernel_size - 1) - channels;
	}

	(module_image + index)[0] = sqrtf(gh*gh + gv * gv);
	orientations[index] = atan2(gv, gh);
}

__global__ void kernel_non_max_suppression(unsigned char* module_image, unsigned char* non_max_image, float* orientations, int width, int height, int weak_color, int strong_color, int low_threshold, int high_threshold)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width <= col || height <= row)
		return;

	int index = row*width + col;

	if (row == 0 || col == 0 || row == height - 1 || col == width - 1)
	{
		non_max_image[index] = module_image[index];
	}
	else
	{
		float angle = orientations[index];
		float r, q;

		if ((0.0 <= angle && angle <= 22.5) || (157.5 <= angle && angle <= 180))
		{
			r = module_image[index + 1];
			q = module_image[index - 1];
		}
		else if (22.5 < angle && angle <= 67.5)
		{
			r = module_image[index + 1 - width];
			q = module_image[index - 1 + width];
		}
		else if (67.5 < angle && angle <= 112.5)
		{
			r = module_image[index - width];
			q = module_image[index + width];
		}
		else
		{
			r = module_image[index - width - 1];
			q = module_image[index + width + 1];
		}

		if (module_image[index] >= r && module_image[index] >= q)
			non_max_image[index] = module_image[index];
		else
			non_max_image[index] = 0;
	}

	if (non_max_image[index] < low_threshold)
		non_max_image[index] = 0;
	else if (non_max_image[index] >= high_threshold)
		non_max_image[index] = strong_color;
	else if (low_threshold <= non_max_image[index] && non_max_image[index] < high_threshold)
		non_max_image[index] = weak_color;
}

__global__ void kernel_hysteresis(unsigned char* non_max_image, unsigned char* filtered_image, int width, int height, int weak_color, int strong_color)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width <= col || height <= row)
		return;

	int index = row * width + col;
	unsigned char* pixel = non_max_image + index;

	if (row == 0 || col == 0 || row == height - 1 || col == width - 1)
	{
		filtered_image[index] = 0;
	}
	else
	{
		if(*pixel == strong_color || (*pixel == weak_color && strong_neighbour(pixel, width, strong_color)))
			filtered_image[index] = strong_color;
		else
			filtered_image[index] = 0;
	}
}

void load_constant_memory_robert_h(float* kernel, int kernel_size)
{
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_robert_kernel_3x3_h), kernel, kernel_size * kernel_size * sizeof(float)));
}

void load_constant_memory_robert_v(float* kernel, int kernel_size)
{
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_robert_kernel_3x3_v), kernel, kernel_size * kernel_size * sizeof(float)));
}

void load_constant_memory_sobel_h(float* kernel, int kernel_size)
{
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_sobel_kernel_3x3_h), kernel, kernel_size * kernel_size * sizeof(float)));
}

void load_constant_memory_sobel_v(float* kernel, int kernel_size)
{
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_sobel_kernel_3x3_v), kernel, kernel_size * kernel_size * sizeof(float)));
}

void load_constant_memory_gaussian(float * kernel, int kernel_size)
{
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_gaussian_kernel_7x7), kernel, kernel_size * kernel_size * sizeof(float)));
}

void naive_robert_convolution_gpu(char* filename, int kernel_size, int kernel_radius, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &filtered_image_size, &f_width, &f_height, kernel_radius);
	filtered_image = (unsigned char*)malloc(filtered_image_size);

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width + block.x - 1) / block.x, (f_height + block.y - 1) / block.y);

		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);

		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_filtered_image, filtered_image_size));
		CHECK(hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice));
		kernel_robert_h_convolution << < grid, block >> > (d_image, d_filtered_image, width, height, channels, kernel_size, kernel_radius);
		CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(filtered_image, d_filtered_image, filtered_image_size, hipMemcpyDeviceToHost));

		end_timer();

		if (output)
			save_file((char*)output_filename_robert[i], filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f %\n\n", speedup());

		hipFree(d_image);
		hipFree(d_filtered_image);
	}
	free(image);
	free(filtered_image);
}

void smem_robert_convolution_gpu(char* filename, int kernel_size, int kernel_radius, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &filtered_image_size, &f_width, &f_height, kernel_radius);
	filtered_image = (unsigned char*)malloc(filtered_image_size);

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width + block.x - 1) / block.x, (f_height + block.y - 1) / block.y);
		int tile_side = block_sizes[i] + kernel_radius * 2;
		size_t tile_size = tile_side * tile_side;

		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);

		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_filtered_image, filtered_image_size));
		CHECK(hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice));

		kernel_robert_h_convolution_smem << < grid, block, tile_size >> > (d_image, d_filtered_image, width, height, channels, tile_side, kernel_size, kernel_radius);

		CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(filtered_image, d_filtered_image, filtered_image_size, hipMemcpyDeviceToHost));

		end_timer();

		if (output)
			save_file((char*)output_filename_robert_smem[i], filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f %\n\n", speedup());

		hipFree(d_image);
		hipFree(d_filtered_image);
	}
	free(image);
	free(filtered_image);
}

void stream_robert_convolution_gpu(char* filename, int kernel_size, int kernel_radius, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &filtered_image_size, &f_width, &f_height, kernel_radius);
	//Pinned memory allocation
	CHECK(hipHostAlloc(&pinned_image, image_size, 0));
	CHECK(hipHostAlloc(&pinned_filtered_image, filtered_image_size, 0));
	memcpy(pinned_image, image, image_size);

	//Chunk_size is the chunk of the input image wich is elaborated by the stream
	size_t chunk_size = (image_size / STREAMS) + width * channels;
	//Chunk_size_result is the chunk of data written by kernels in the output
	size_t chunk_size_result = filtered_image_size / STREAMS;

	//Stream creation
	hipStream_t stream[STREAMS];
	for (int i = 0; i < STREAMS; i++)
		CHECK(hipStreamCreate(&stream[i]));

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width + block.x - 1) / block.x, ((f_height / STREAMS) + block.y - 1) / block.y);

		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);
		printf("Streams: %d\n", STREAMS);
		//Offset_input is the offset from which a kernel starts to read input image data
		int offset_input = 0;
		//Since the input potentially has more channels than the output(the output is always in grayscale), we need a different offset.
		int offset_output = 0;

		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_filtered_image, filtered_image_size));
		for (int j = 0; j < STREAMS; j++)
		{
			CHECK(hipMemcpyAsync(&d_image[offset_input], &pinned_image[offset_input], chunk_size, hipMemcpyHostToDevice, stream[j]));
			kernel_robert_h_convolution_stream << <grid, block, 0, stream[j] >> > (d_image, d_filtered_image, width, height / STREAMS, channels, offset_input, offset_output, kernel_size, kernel_radius);
			CHECK(hipMemcpyAsync(&pinned_filtered_image[offset_output], &d_filtered_image[offset_output], chunk_size_result, hipMemcpyDeviceToHost, stream[j]));
			offset_input += (int)((image_size / STREAMS) - width * channels);
			offset_output += (int)chunk_size_result;
		}

		for (int j = 0; j < STREAMS; j++)
			CHECK(hipStreamSynchronize(stream[j]));
		end_timer();

		if (output)
			save_file((char*)output_filename_robert_stream[i], pinned_filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f %\n\n", speedup());

		hipFree(d_image);
		hipFree(d_filtered_image);
	}


	for (int i = 0; i < STREAMS; i++)
		hipStreamDestroy(stream[i]);

	free(image);
	hipHostFree(pinned_image);
	hipHostFree(pinned_filtered_image);
}

void naive_sobel_module_gpu(char * filename, int kernel_size, int kernel_radius, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &filtered_image_size, &f_width, &f_height, kernel_radius);
	filtered_image = (unsigned char*)malloc(filtered_image_size);

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width + block.x - 1) / block.x, (f_height + block.y - 1) / block.y);

		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);

		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_filtered_image, filtered_image_size));
		CHECK(hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice));
		kernel_module_sobel << < grid, block >> > (d_image, d_filtered_image, width, height, channels, kernel_size, kernel_radius);
		CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(filtered_image, d_filtered_image, filtered_image_size, hipMemcpyDeviceToHost));

		end_timer();

		if (output)
			save_file((char*)output_filename_module[i], filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f %\n\n", speedup());

		hipFree(d_image);
		hipFree(d_filtered_image);
	}
	free(image);
	free(filtered_image);
}

void smem_sobel_module_gpu(char * filename, int kernel_size, int kernel_radius, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &filtered_image_size, &f_width, &f_height, kernel_radius);
	filtered_image = (unsigned char*)malloc(filtered_image_size);

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width + block.x - 1) / block.x, (f_height + block.y - 1) / block.y);
		int tile_side = block_sizes[i] + kernel_radius * 2;
		size_t tile_size = tile_side * tile_side;

		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);

		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_filtered_image, filtered_image_size));
		CHECK(hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice));

		kernel_module_sobel_smem << < grid, block, tile_size >> > (d_image, d_filtered_image, width, height, channels, tile_side, kernel_size, kernel_radius);

		CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(filtered_image, d_filtered_image, filtered_image_size, hipMemcpyDeviceToHost));

		end_timer();

		if (output)
			save_file((char*)output_filename_module_smem[i], filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f %\n\n", speedup());

		hipFree(d_image);
		hipFree(d_filtered_image);
	}
	free(image);
	free(filtered_image);
}

void stream_sobel_module_gpu(char * filename, int kernel_size, int kernel_radius, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &filtered_image_size, &f_width, &f_height, kernel_radius);
	//Pinned memory allocation
	CHECK(hipHostAlloc(&pinned_image, image_size, 0));
	CHECK(hipHostAlloc(&pinned_filtered_image, filtered_image_size, 0));
	memcpy(pinned_image, image, image_size);

	//Chunk_size is the chunk of the input image wich is elaborated by the stream
	size_t chunk_size = (image_size / STREAMS) + width * channels;
	//Chunk_size_result is the chunk of data written by kernels in the output
	size_t chunk_size_result = filtered_image_size / STREAMS;

	//Stream creation
	hipStream_t stream[STREAMS];
	for (int i = 0; i < STREAMS; i++)
		CHECK(hipStreamCreate(&stream[i]));

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width + block.x - 1) / block.x, ((f_height / STREAMS) + block.y - 1) / block.y);

		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);
		printf("Streams: %d\n", STREAMS);
		//Offset_input is the offset from which a kernel starts to read input image data
		int offset_input = 0;
		//Since the input potentially has more channels than the output(the output is always in grayscale), we need a different offset.
		int offset_output = 0;

		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_filtered_image, filtered_image_size));
		for (int j = 0; j < STREAMS; j++)
		{
			CHECK(hipMemcpyAsync(&d_image[offset_input], &pinned_image[offset_input], chunk_size, hipMemcpyHostToDevice, stream[j]));
			kernel_module_sobel_stream << <grid, block, 0, stream[j] >> > (d_image, d_filtered_image, width, height / STREAMS, channels, offset_input, offset_output, kernel_size, kernel_radius);
			CHECK(hipMemcpyAsync(&pinned_filtered_image[offset_output], &d_filtered_image[offset_output], chunk_size_result, hipMemcpyDeviceToHost, stream[j]));
			offset_input += (int)((image_size / STREAMS) - width * channels);
			offset_output += (int)chunk_size_result;
		}

		for (int j = 0; j < STREAMS; j++)
			CHECK(hipStreamSynchronize(stream[j]));
		end_timer();

		if (output)
			save_file((char*)output_filename_module_stream[i], pinned_filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f %\n\n", speedup());

		hipFree(d_image);
		hipFree(d_filtered_image);
	}


	for (int i = 0; i < STREAMS; i++)
		hipStreamDestroy(stream[i]);

	free(image);
	hipHostFree(pinned_image);
	hipHostFree(pinned_filtered_image);
}

void naive_canny_gpu(char * filename, float * kernel_h, float * kernel_v, float * gaussian_kernel, float sigma, int kernel_size, int kernel_radius, float low_threshold_ratio, float high_threshold_ratio, bool output)
{
	int sobel_kernel_size = 3;
	int sobel_kernel_radius = 1;
	size_t gaussian_image_size;
	int f_width_gaussian, f_height_gaussian;
	image = load_file_details(filename, &width, &height, &channels, &image_size, &gaussian_image_size, &f_width_gaussian, &f_height_gaussian, kernel_radius);

	f_width = f_width_gaussian - sobel_kernel_radius * 2;
	f_height = f_height_gaussian - sobel_kernel_radius * 2;

	size_t module_image_size = f_width * f_height;
	filtered_image_size = f_width * f_height;

	size_t orientations_size = sizeof(float) * f_width*f_height;
	filtered_image = (unsigned char*)malloc(filtered_image_size);

	int strong_color = 255;
	int weak_color = 40;
	float high_threshold = high_threshold_ratio * strong_color;
	float low_threshold = low_threshold_ratio * high_threshold;

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((width + block.x - 1) / block.x, (height + block.y - 1) / block.y);
		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);
		begin_timer();
		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_gaussian_image, gaussian_image_size));
		CHECK(hipMalloc((void**)&d_module_image, filtered_image_size));
		CHECK(hipMalloc((void**)&d_non_max_image, filtered_image_size));
		CHECK(hipMalloc((void**)&d_orientations, orientations_size));
		CHECK(hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice));

		//Gaussian filter
		kernel_gaussian_convolution << <block, grid >> > (d_image, d_gaussian_image, width, height, channels, kernel_size, kernel_radius);
		grid = dim3((f_width + block.x - 1) / block.x, (f_height + block.y - 1) / block.y);
		//Module and orientations
		kernel_module_orientation << <block, grid >> > (d_gaussian_image, d_module_image, d_orientations, f_width_gaussian, f_height_gaussian, 1, sobel_kernel_size, sobel_kernel_radius);
		grid = dim3((f_width + block.x - 3) / block.x, (f_height + block.y - 3) / block.y);
		//Non max suppression
		kernel_non_max_suppression << <block, grid >> > (d_module_image, d_non_max_image, d_orientations, f_width, f_height, weak_color, strong_color, low_threshold, high_threshold);
		//Hysteresis
		kernel_hysteresis << <block, grid >> > (d_non_max_image, d_module_image, f_width, f_height, weak_color, strong_color);
		CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(filtered_image, d_module_image, filtered_image_size, hipMemcpyDeviceToHost));

		end_timer();

		if (output)
			save_file((char*)output_filename_canny[i], filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f %\n\n", speedup());

		CHECK(hipFree(d_image));
		CHECK(hipFree(d_gaussian_image));
		CHECK(hipFree(d_module_image));
		CHECK(hipFree(d_non_max_image));
		CHECK(hipFree(d_orientations));
	}

	free(image);
	free(filtered_image);
}
