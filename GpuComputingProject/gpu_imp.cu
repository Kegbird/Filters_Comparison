#include "hip/hip_runtime.h"
﻿#include"gpu_imp.cuh"
#include "utils.h"
#include <stdio.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""

#define STREAMS 8
#define BLOCK_SIZES 2

#define GAUSS_KERNEL_CODE 0
#define SOBEL_KERNEL_CODE_H 1
#define SOBEL_KERNEL_CODE_V 2
#define ROBERT_KERNEL_CODE_H 3
#define ROBERT_KERNEL_CODE_V 3

__constant__ float d_robert_kernel_3x3_h[3][3];
__constant__ float d_robert_kernel_3x3_v[3][3];

__constant__ float d_sobel_kernel_3x3_h[3][3];
__constant__ float d_sobel_kernel_3x3_v[3][3];

__constant__ float d_gaussian_kernel_7x7[7][7];

const int block_sizes[BLOCK_SIZES] = { 16, 32 };

unsigned char* image;
unsigned char* filtered_image;
unsigned char* pinned_image;
unsigned char* pinned_filtered_image;
unsigned char* d_image;
unsigned char* d_filtered_image;
unsigned char* d_gaussian_image;
unsigned char* d_module_image;
unsigned char* d_non_max_image;
float* d_orientations;

int width;
int height;
int f_width;
int f_height;
int f_width_gaussian;
int f_height_gaussian;
int channels;
size_t image_size;
size_t filtered_image_size;
size_t gaussian_image_size;
size_t orientations_size;

const char* output_filename_robert[] = { "Conv_Robert_Naive_16x16.png",
									"Conv_Robert_Naive_32x32.png" };

const char* output_filename_robert_smem[] = { "Conv_Robert_Smem_16x16.png",
									"Conv_Robert_Smem_32x32.png" };

const char* output_filename_robert_stream[] = { "Conv_Robert_Stream_16x16.png",
									"Conv_Robert_Stream_32x32.png" };

const char* output_filename_robert_stream_smem[] = { "Conv_Robert_Smem_Stream_16x16.png",
									"Conv_Robert_Smem_Stream_32x32.png" };

const char* output_filename_module[] = { "Module_Naive_16x16.png",
									"Module_Naive_32x32.png" };

const char* output_filename_module_smem[] = { "Module_Smem_16x16.png",
									"Module_Smem_32x32.png" };

const char* output_filename_module_stream[] = { "Module_Stream_16x16.png",
									"Module_Stream_32x32.png" };

const char* output_filename_module_stream_smem[] = { "Module_Stream_Smem_16x16.png",
									"Module_Stream_Smem_32x32.png" };

const char* output_filename_canny[] = { "Canny_Naive_16x16.png",
										"Canny_Naive_32x32.png" };

const char* output_filename_canny_smem[] = { "Canny_Smem_16x16.png",
										"Canny_Smem_32x32.png" };

const char* output_filename_canny_stream[] = { "Canny_Stream_16x16.png",
										"Canny_Stream_32x32.png" };

const char* output_filename_canny_stream_smem[] = { "Canny_Stream_Smem_16x16.png",
													"Canny_Stream_Smem_32x32.png" };

void load_constant_memory_robert_h(float* kernel, int kernel_side)
{
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_robert_kernel_3x3_h), kernel, kernel_side * kernel_side * sizeof(float)));
}

void load_constant_memory_robert_v(float* kernel, int kernel_side)
{
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_robert_kernel_3x3_v), kernel, kernel_side * kernel_side * sizeof(float)));
}

void load_constant_memory_sobel_h(float* kernel, int kernel_side)
{
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_sobel_kernel_3x3_h), kernel, kernel_side * kernel_side * sizeof(float)));
}

void load_constant_memory_sobel_v(float* kernel, int kernel_side)
{
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_sobel_kernel_3x3_v), kernel, kernel_side * kernel_side * sizeof(float)));
}

void load_constant_memory_gaussian(float * kernel, int kernel_side)
{
	CHECK(hipMemcpyToSymbol(HIP_SYMBOL(d_gaussian_kernel_7x7), kernel, kernel_side * kernel_side * sizeof(float)));
}

__device__ float device_grayscale(unsigned char* pixel, int channels)
{
	float color = 0;
	for (int j = 0; j < channels; j++)
		color += pixel[j] / channels;
	return color;
}

__device__ void device_fill_shared_memory_tile(unsigned char* pixel, unsigned char* image_tile, int width, int height, int channels, int tile_side, int tile_index, int row, int col, int kernel_radius)
{
	image_tile[tile_index] = device_grayscale(pixel, channels);

	if ((threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1) || (row == height - (kernel_radius) * 2 - 1) || (col == width - (kernel_radius) * 2 - 1))
	{
		//Bottom right corner thread
		for (int i = 1; i <= kernel_radius * 2; i++)
		{
			image_tile[tile_index + i] = device_grayscale(pixel + channels * i, channels);
			image_tile[tile_index + tile_side * i] = device_grayscale(pixel + width * channels * i, channels);

			for (int j = 1; j <= kernel_radius * 2; j++)
				image_tile[tile_index + tile_side * i + j] = device_grayscale(pixel + width * channels * i + channels * j, channels);
		}
	}
	else if (threadIdx.x == blockDim.x - 1 || (col == width - (kernel_radius) * 2 - 1))
	{
		//Right edge thread
		for (int i = 1; i <= kernel_radius * 2; i++)
			image_tile[tile_index + i] = device_grayscale(pixel + channels * i, channels);
	}
	else if (threadIdx.y == blockDim.y - 1 || (row == height - (kernel_radius) * 2 - 1))
	{
		//Bottom left corner thread
		for (int i = 1; i <= kernel_radius * 2; i++)
			image_tile[tile_index + tile_side * i] = device_grayscale(pixel + width * channels * i, channels);
	}

}

__device__ void device_fill_shared_memory_tile_as_frame(unsigned char* pixel, unsigned char* image_tile, int index, int tile_side, int tile_index, int row, int col, int width, int height)
{
	if (row == 0 || col == 0 || row == height - 1 || col == width - 1)
	{
		//Image corners
		image_tile[tile_index] = *(pixel + index);
	}
	else if (threadIdx.x == 0 && threadIdx.y == 0)
	{
		//Filling block top left corner
		image_tile[0] = *(pixel + index - width - 1);
		image_tile[1] = *(pixel + index - width);
		image_tile[tile_side - 1] = *(pixel + index - 1);
		image_tile[tile_index] = *(pixel + index);
	}
	else if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0)
	{
		//Filling top right corner
		image_tile[tile_index - tile_side] = *(pixel + index - width);
		image_tile[tile_index - tile_side + 1] = *(pixel + index - width + 1);
		image_tile[tile_index] = *(pixel + index);
		image_tile[tile_index + 1] = *(pixel + index + 1);
	}
	else if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1)
	{
		//Filling bottom left
		image_tile[tile_index - 1] = *(pixel + index - 1);
		image_tile[tile_index] = *(pixel + index);
		image_tile[tile_index + tile_side - 1] = *(pixel + index + width - 1);
		image_tile[tile_index + tile_side] = *(pixel + index + width);

	}
	else if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.x - 1)
	{
		//Filling bottom right
		image_tile[tile_index] = *(pixel + index);
		image_tile[tile_index + 1] = *(pixel + index + 1);
		image_tile[tile_index + tile_side] = *(pixel + index + width);
		image_tile[tile_index + tile_side + 1] = *(pixel + index + width + 1);
	}
	else if (threadIdx.y == 0)
	{
		//Top edge
		image_tile[tile_index - tile_side] = *(pixel + index - width);
		image_tile[tile_index] = *(pixel + index);
	}
	else if (threadIdx.x == 0)
	{
		//Left edge
		image_tile[tile_index - 1] = *(pixel + index - 1);
		image_tile[tile_index] = *(pixel + index);
	}
	else if (threadIdx.x == blockDim.x - 1)
	{
		//Right edge
		image_tile[tile_index] = *(pixel + index);
		image_tile[tile_index + 1] = *(pixel + index + 1);
	}
	else
	{
		//Bottom edge
		image_tile[tile_index] = *(pixel + index);
		image_tile[tile_index + tile_side] = *(pixel + index + width);
	}
}

__device__ float device_convolution(unsigned char* pixel, int channels, int width, float* kernel, int kernel_side, int kernel_radius)
{
	float result = 0;
	for (int i = 0; i < kernel_side; i++)
	{
		for (int j = 0; j < kernel_side; j++)
		{
			result += device_grayscale(pixel, channels) * kernel[i*kernel_side + j];
			pixel += channels;
		}
		pixel += (width * channels) - channels * (kernel_side - 1) - channels;
	}
	if (result < 0)
		result = 0;
	return result;
}

__device__ float device_convolution_smem(float* kernel, unsigned char* image_tile, int tile_index, int tile_side, int kernel_side, int kernel_radius)
{
	float result = 0.0;
	for (int i = 0; i < kernel_side; i++)
	{
		for (int j = 0; j < kernel_side; j++, tile_index++)
			result += image_tile[tile_index] * kernel[i*kernel_side + j];
		tile_index += tile_side - kernel_radius * 2 - 1;
	}
	if (result < 0)
		result = 0;
	return result;
}

__device__ bool device_strong_neighbour(unsigned char* pixel, int width, int strong_color)
{
	if (*(pixel - width - 1) == strong_color || *(pixel - width) == strong_color || *(pixel - width + 1) == strong_color ||
		*(pixel - 1) == strong_color || *(pixel + 1) == strong_color ||
		*(pixel + width - 1) == strong_color || *(pixel + width) == strong_color || *(pixel + width + 1) == strong_color)
		return true;
	return false;
}

__device__ float device_module(unsigned char* pixel, int channels, int width)
{
	int kernel_side = 3;
	float gh = 0.0, gv = 0.0;
	for (int i = 0; i < kernel_side; i++)
	{
		//Evaluating gh and gv
		for (int j = 0; j < kernel_side; j++, pixel += channels)
		{
			gh += device_grayscale(pixel, channels) * d_sobel_kernel_3x3_h[i][j];
			gv += device_grayscale(pixel, channels) * d_sobel_kernel_3x3_v[i][j];
		}
		pixel += (width * channels) - channels * (kernel_side - 1) - channels;
	}

	return sqrtf(gh*gh + gv * gv);
}

__device__ float device_module_smem(unsigned char* image_tile, int tile_index, int tile_side)
{
	float gh = 0, gv = 0;
	for (int i = 0; i < 3; i++)
	{
		for (int j = 0; j < 3; j++, tile_index++)
		{
			gh += image_tile[tile_index] * d_sobel_kernel_3x3_h[i][j];
			gv += image_tile[tile_index] * d_sobel_kernel_3x3_v[i][j];
		}
		tile_index += tile_side - 3;
	}
	return sqrtf(gh*gh + gv * gv);
}

__device__ float device_module(unsigned char* pixel, int channels, int width, float* gh, float* gv)
{
	for (int i = 0; i < 3; i++)
	{
		//Evaluating gh and gv
		for (int j = 0; j < 3; j++)
		{
			*gh += *pixel * d_sobel_kernel_3x3_h[i][j];
			*gv += *pixel * d_sobel_kernel_3x3_v[i][j];
			pixel += channels;
		}
		pixel += (width * channels) - channels * 2 - channels;
	}
	return sqrtf((*gh)*(*gh) + (*gv)*(*gv));
}

__device__ void device_non_max_suppression(unsigned char* non_max_image, unsigned char* module_image, float* orientations, int index, int width, int height, int row, int col)
{
	if (row == 0 || col == 0 || row == height - 1 || col == width - 1)
	{
		non_max_image[index] = module_image[index];
	}
	else
	{
		float angle = orientations[index];
		int r, q;

		if ((0.0 <= angle && angle <= 22.5) || (157.5 <= angle && angle <= 180))
		{
			r = module_image[index + 1];
			q = module_image[index - 1];
		}
		else if (22.5 < angle && angle <= 67.5)
		{
			r = module_image[index + 1 - width];
			q = module_image[index - 1 + width];
		}
		else if (67.5 < angle && angle <= 112.5)
		{
			r = module_image[index - width];
			q = module_image[index + width];
		}
		else
		{
			r = module_image[index - width - 1];
			q = module_image[index + width + 1];
		}

		if (module_image[index] >= r && module_image[index] >= q)
			non_max_image[index] = module_image[index];
		else
			non_max_image[index] = 0;
	}
}

__device__ void device_non_max_suppression_smem(unsigned char* non_max_image, unsigned char* image_tile, int tile_index, int tile_side, float* orientations, int index, int row, int col, int width, int height)
{
	if (row == 0 || col == 0 || row == height - 1 || col == width - 1)
	{
		non_max_image[index] = image_tile[tile_index];
	}
	else
	{
		float angle = orientations[index];
		int r, q;

		if ((0.0 <= angle && angle <= 22.5) || (157.5 <= angle && angle <= 180))
		{
			r = image_tile[tile_index + 1];
			q = image_tile[tile_index - 1];
		}
		else if (22.5 < angle && angle <= 67.5)
		{
			r = image_tile[tile_index + 1 - tile_side];
			q = image_tile[tile_index - 1 + tile_side];
		}
		else if (67.5 < angle && angle <= 112.5)
		{
			r = image_tile[tile_index - tile_side];
			q = image_tile[tile_index + tile_side];
		}
		else
		{
			r = image_tile[tile_index - tile_side - 1];
			q = image_tile[tile_index + tile_side + 1];
		}

		if (image_tile[tile_index] >= r && image_tile[tile_index] >= q)
		{
			non_max_image[index] = image_tile[tile_index];
		}
		else
		{
			image_tile[tile_index] = 0;
			non_max_image[index] = 0;
		}
	}
}

__device__ float* pick_kernel(int kernel_code)
{
	switch (kernel_code)
	{
	case GAUSS_KERNEL_CODE:
		return &d_gaussian_kernel_7x7[0][0];
		break;
	case SOBEL_KERNEL_CODE_H:
		return &d_sobel_kernel_3x3_h[0][0];
		break;
	case SOBEL_KERNEL_CODE_V:
		return &d_sobel_kernel_3x3_v[0][0];
		break;
	case ROBERT_KERNEL_CODE_H:
		return &d_robert_kernel_3x3_h[0][0];
		break;
	default:
		return &d_robert_kernel_3x3_v[0][0];
		break;
	}
}

__global__ void kernel_convolution(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels, int kernel_side, int kernel_radius, int kernel_code)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - (kernel_radius) * 2 <= col || height - (kernel_radius) * 2 <= row)
		return;

	int index = row * width + col - ((kernel_radius) * 2)*row;
	unsigned char* pixel = image + row * width * channels + col * channels;

	*(filtered_image + index) = device_convolution(pixel, channels, width, pick_kernel(kernel_code), kernel_side, kernel_radius);
}

__global__ void kernel_convolution_smem(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels, int tile_side, int kernel_side, int kernel_radius, int kernel_code)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if ((width - (kernel_radius) * 2 <= col || height - (kernel_radius) * 2 <= row))
		return;

	extern __shared__ unsigned char image_tile[];

	unsigned char *pixel = image + row * width *channels + col * channels;

	int tile_index = threadIdx.y*tile_side + threadIdx.x;

	device_fill_shared_memory_tile(pixel, image_tile, width, height, channels, tile_side, tile_index, row, col, kernel_radius);

	__syncthreads();

	float *kernel = pick_kernel(kernel_code);

	int index = row * width + col - ((kernel_radius) * 2)*row;

	(filtered_image + index)[0] = device_convolution_smem(kernel, image_tile, tile_index, tile_side, kernel_side, kernel_radius);
}

__global__ void kernel_convolution_stream(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels, int row_offset, int image_offset, int filtered_image_offset, int kernel_side, int kernel_radius, int kernel_code)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - (kernel_radius) * 2 <= col || height - (kernel_radius) * 2 <= row + row_offset)
		return;

	unsigned char* pixel = image + (row * width * channels + col * channels) + image_offset;

	int index = row * (width - kernel_radius * 2) + col + filtered_image_offset;

	*(filtered_image + index) = device_convolution(pixel, channels, width, pick_kernel(kernel_code), kernel_side, kernel_radius);
}

__global__ void kernel_convolution_stream_smem(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels, int tile_side, int row_offset, int image_offset, int filtered_image_offset, int kernel_side, int kernel_radius, int kernel_code)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - (kernel_radius) * 2 <= col || height - (kernel_radius) * 2 <= row + row_offset)
		return;

	extern __shared__ unsigned char image_tile[];

	unsigned char* pixel = image + (row * width * channels + col * channels) + image_offset;

	int tile_index = threadIdx.y*tile_side + threadIdx.x;

	device_fill_shared_memory_tile(pixel, image_tile, width, height, channels, tile_side, tile_index, row + row_offset, col, kernel_radius);

	__syncthreads();

	float *kernel = pick_kernel(kernel_code);

	int index = (row * (width - kernel_radius * 2) + col) + filtered_image_offset;
	(filtered_image + index)[0] = device_convolution_smem(kernel, image_tile, tile_index, tile_side, kernel_side, kernel_radius);
}

__global__ void kernel_module(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - 2 <= col || height - 2 <= row)
		return;

	int index = row * width + col - 2 * row;
	unsigned char* pixel = image + row * width * channels + col * channels;
	(filtered_image + index)[0] = device_module(pixel, channels, width);
}

__global__ void kernel_module_smem(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels, int tile_side)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if ((width - 2 <= col || height - 2 <= row))
		return;

	extern __shared__ unsigned char image_tile[];

	unsigned char *pixel = image + row * width *channels + col * channels;

	int tile_index = threadIdx.y*tile_side + threadIdx.x;

	device_fill_shared_memory_tile(pixel, image_tile, width, height, channels, tile_side, tile_index, row, col, 1);

	__syncthreads();

	int index = row * width + col - 2 * row;

	(filtered_image + index)[0] = device_module_smem(image_tile, tile_index, tile_side);
}

__global__ void kernel_module_stream(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels, int row_offset, int image_offset, int filtered_image_offset)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - 2 <= col || height - 2 <= row + row_offset)
		return;

	unsigned char* pixel = image + row * width * channels + col * channels + image_offset;
	int index = filtered_image_offset + (row * width + col - 2 * row);
	(filtered_image + index)[0] = device_module(pixel, channels, width);
}

__global__ void kernel_module_stream_smem(unsigned char* image, unsigned char* filtered_image, int width, int height, int channels, int tile_side, int row_offset, int image_offset, int filtered_image_offset)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - 2 <= col || height - 2 <= row + row_offset)
		return;

	extern __shared__ unsigned char image_tile[];

	unsigned char* pixel = image + (row * width * channels + col * channels) + image_offset;

	int tile_index = threadIdx.y*tile_side + threadIdx.x;

	device_fill_shared_memory_tile(pixel, image_tile, width, height, channels, tile_side, tile_index, row + row_offset, col, 1);

	__syncthreads();

	int index = filtered_image_offset + (row * width + col - 2 * row);

	(filtered_image + index)[0] = device_module_smem(image_tile, tile_index, tile_side);
}

__global__ void kernel_module_orientation(unsigned char* gaussian_filtered_image, unsigned char* module_image, float* orientations, int width, int height, int channels)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - 2 <= col || height - 2 <= row)
		return;

	unsigned char* pixel = gaussian_filtered_image + row * width * channels + col * channels;

	float gh = 0.0, gv = 0.0;
	int index = row * width + col - 2 * row;
	(module_image + index)[0] = device_module(pixel, channels, width, &gh, &gv);
	orientations[index] = atan2(gv, gh);
}

__global__ void kernel_module_orientation_smem(unsigned char* gaussian_filtered_image, unsigned char* module_image, float* orientations, int width, int height, int channels, int tile_side)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if ((width - 2 <= col || height - 2 <= row))
		return;

	extern __shared__ unsigned char image_tile[];
	unsigned char *pixel = gaussian_filtered_image + row * width *channels + col * channels;

	int tile_index = threadIdx.y*tile_side + threadIdx.x;

	device_fill_shared_memory_tile(pixel, image_tile, width, height, channels, tile_side, tile_index, row, col, 1);
	
	__syncthreads();

	float gh = 0, gv = 0;
	int index = row * width + col - 2 * row;
	(module_image + index)[0] = device_module(pixel, channels, width, &gh, &gv);
	orientations[index] = atan2(gv, gh);
}

__global__ void kernel_module_orientation_stream(unsigned char* gaussian_filtered_image, unsigned char* module_image, float* orientations, int width, int height, int channels, int row_offset, int image_offset, int filtered_image_offset)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width - 2 <= col || height - 2 <= row + row_offset)
		return;

	unsigned char* pixel = gaussian_filtered_image + row * width * channels + col * channels + image_offset;

	float gh = 0.0, gv = 0.0;
	int index = filtered_image_offset + (row * width + col - 2 * row);
	(module_image + index)[0] = device_module(pixel, channels, width, &gh, &gv);
	orientations[index] = atan2(gv, gh);
}

__global__ void kernel_non_max_suppression(unsigned char* module_image, unsigned char* non_max_image, float* orientations, int width, int height, int weak_color, int strong_color, float low_threshold, float high_threshold)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width <= col || height <= row)
		return;

	int index = row * width + col;

	device_non_max_suppression(non_max_image, module_image, orientations, index, width, height, row, col);

	if (non_max_image[index] < low_threshold)
		non_max_image[index] = 0;
	else if (non_max_image[index] >= high_threshold)
		non_max_image[index] = strong_color;
	else if (low_threshold <= non_max_image[index] && non_max_image[index] < high_threshold)
		non_max_image[index] = weak_color;
}

__global__ void kernel_non_max_suppression_smem(unsigned char* module_image, unsigned char* non_max_image, float* orientations, int width, int height, int weak_color, int strong_color, float low_threshold, float high_threshold, int tile_side)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width <= col || height <= row)
		return;

	extern __shared__ unsigned char image_tile[];

	int index = row * width + col;

	int tile_index = (threadIdx.y + 1)*tile_side + threadIdx.x + 1;

	device_fill_shared_memory_tile_as_frame(module_image, image_tile, index, tile_side, tile_index, row, col, width, height);

	__syncthreads();

	device_non_max_suppression_smem(non_max_image, image_tile, tile_index, tile_side, orientations, index, row, col, width, height);

	if (image_tile[tile_index] < low_threshold)
		non_max_image[index] = 0;
	else if (image_tile[tile_index] >= high_threshold)
		non_max_image[index] = strong_color;
	else if (low_threshold <= image_tile[tile_index] && image_tile[tile_index] < high_threshold)
		non_max_image[index] = weak_color;
}

__global__ void kernel_non_max_suppression_stream(unsigned char* module_image, unsigned char* non_max_image, float* orientations, int width, int height, int row_offset, int image_offset, int weak_color, int strong_color, float low_threshold, float high_threshold)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width <= col || height <= row + row_offset)
		return;

	int index = row * width + col + image_offset;

	device_non_max_suppression(non_max_image, module_image, orientations, index, width, height, row + row_offset, col);

	if (non_max_image[index] < low_threshold)
		non_max_image[index] = 0;
	else if (non_max_image[index] >= high_threshold)
		non_max_image[index] = strong_color;
	else if (low_threshold <= non_max_image[index] && non_max_image[index] < high_threshold)
		non_max_image[index] = weak_color;
}

__global__ void kernel_hysteresis(unsigned char* non_max_image, unsigned char* filtered_image, int width, int height, int weak_color, int strong_color)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width <= col || height <= row)
		return;

	int index = row * width + col;

	if (row == 0 || col == 0 || row == height - 1 || col == width - 1)
	{
		filtered_image[index] = 0;
	}
	else
	{
		unsigned char* pixel = non_max_image + index;
		if (*pixel == strong_color || (*pixel == weak_color && device_strong_neighbour(pixel, width, strong_color)))
			filtered_image[index] = strong_color;
		else
			filtered_image[index] = 0;
	}
}

__global__ void kernel_hysteresis_smem(unsigned char* non_max_image, unsigned char* filtered_image, int width, int height, int weak_color, int strong_color, int tile_side)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width <= col || height <= row)
		return;

	int index = row * width + col;
	extern __shared__ unsigned char image_tile[];

	int tile_index = (threadIdx.y + 1)*tile_side + threadIdx.x + 1;

	device_fill_shared_memory_tile_as_frame(non_max_image, image_tile, index, tile_side, tile_index, row, col, width, height);

	__syncthreads();

	if (row == 0 || col == 0 || row == height - 1 || col == width - 1)
	{
		filtered_image[index] = 0;
	}
	else
	{
		unsigned char* pixel = image_tile + tile_index;
		if (*pixel == strong_color || (*pixel == weak_color && device_strong_neighbour(pixel, tile_side, strong_color)))
			filtered_image[index] = strong_color;
		else
			filtered_image[index] = 0;
	}
}

__global__ void kernel_hysteresis_stream(unsigned char* non_max_image, unsigned char* filtered_image, int width, int height, int row_offset, int image_offset, int weak_color, int strong_color)
{
	int row = threadIdx.y + blockIdx.y*blockDim.y;
	int col = threadIdx.x + blockIdx.x*blockDim.x;

	if (width <= col || height <= row + row_offset)
		return;

	int index = row * width + col + image_offset;

	if (row + row_offset == 0 || col == 0 || row + row_offset == height - 1 || col == width - 1)
	{
		filtered_image[index] = 0;
	}
	else
	{
		unsigned char* pixel = non_max_image + index;
		if (*pixel == strong_color || (*pixel == weak_color && device_strong_neighbour(pixel, width, strong_color)))
			filtered_image[index] = strong_color;
		else
			filtered_image[index] = 0;
	}
}

void naive_robert_convolution_gpu(const char* filename, int kernel_side, int kernel_radius, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &filtered_image_size, &f_width, &f_height, kernel_radius);
	filtered_image = (unsigned char*)malloc(filtered_image_size);

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width + block.x - 1) / block.x, (f_height + block.y - 1) / block.y);

		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);

		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_filtered_image, filtered_image_size));
		CHECK(hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice));
		kernel_convolution << <grid, block >> > (d_image, d_filtered_image, width, height, channels, kernel_side, kernel_radius, ROBERT_KERNEL_CODE_H);
		CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(filtered_image, d_filtered_image, filtered_image_size, hipMemcpyDeviceToHost));

		end_timer();

		if (output)
			save_file(output_filename_robert[i], filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f\n\n", speedup());

		hipFree(d_image);
		hipFree(d_filtered_image);
	}
	free(image);
	free(filtered_image);
}

void smem_robert_convolution_gpu(const char* filename, int kernel_side, int kernel_radius, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &filtered_image_size, &f_width, &f_height, kernel_radius);
	filtered_image = (unsigned char*)malloc(filtered_image_size);

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width + block.x - 1) / block.x, (f_height + block.y - 1) / block.y);
		int tile_side = block_sizes[i] + kernel_radius * 2;
		size_t tile_size = tile_side * tile_side;

		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);

		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_filtered_image, filtered_image_size));
		CHECK(hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice));

		kernel_convolution_smem << < grid, block, tile_size >> > (d_image, d_filtered_image, width, height, channels, tile_side, kernel_side, kernel_radius, ROBERT_KERNEL_CODE_H);

		CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(filtered_image, d_filtered_image, filtered_image_size, hipMemcpyDeviceToHost));

		end_timer();

		if (output)
			save_file(output_filename_robert_smem[i], filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f\n\n", speedup());

		hipFree(d_image);
		hipFree(d_filtered_image);
	}
	free(image);
	free(filtered_image);
}

void stream_robert_convolution_gpu(const char* filename, int kernel_side, int kernel_radius, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &filtered_image_size, &f_width, &f_height, kernel_radius);
	//Pinned memory allocation
	CHECK(hipHostAlloc(&pinned_image, image_size, 0));
	CHECK(hipHostAlloc(&pinned_filtered_image, filtered_image_size, 0));
	memcpy(pinned_image, image, image_size);

	//Chunk_size is the chunk of the input image wich is elaborated by the stream
	size_t chunk_size = image_size / STREAMS;
	//Chunk_size_result is the chunk of data written by kernels in the output
	size_t chunk_size_result = filtered_image_size / STREAMS;

	//Stream creation
	hipStream_t stream[STREAMS];
	for (int i = 0; i < STREAMS; i++)
		CHECK(hipStreamCreate(&stream[i]));

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width + block.x - 1) / block.x, ((f_height / STREAMS) + block.y - 1) / block.y);

		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);
		printf("Streams: %d\n", STREAMS);
		//Offset_input is the offset from which a kernel starts to read input image data
		int offset_input = 0;
		//Since the input potentially has more channels than the output(the output is always in grayscale), we need a different offset.
		int offset_output = 0;
		int row_offset;
		int image_offset;
		int filtered_image_offset;

		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_filtered_image, filtered_image_size));
		for (int j = 0; j < STREAMS; j++)
		{
			row_offset = j * (height / STREAMS);
			image_offset = j * width*(height / STREAMS)*channels;
			filtered_image_offset = j * (width - kernel_radius * 2)*(height / STREAMS);
			CHECK(hipMemcpyAsync(&d_image[offset_input], &pinned_image[offset_input], chunk_size, hipMemcpyHostToDevice, stream[j]));
			kernel_convolution_stream << <grid, block, 0, stream[j] >> > (d_image, d_filtered_image, width, height, channels, row_offset, image_offset, filtered_image_offset, kernel_side, kernel_radius, ROBERT_KERNEL_CODE_H);
			CHECK(hipMemcpyAsync(&pinned_filtered_image[offset_output], &d_filtered_image[offset_output], chunk_size_result, hipMemcpyDeviceToHost, stream[j]));
			offset_input += (int)chunk_size;
			offset_output += (int)chunk_size_result;
		}

		for (int j = 0; j < STREAMS; j++)
			CHECK(hipStreamSynchronize(stream[j]));
		end_timer();

		if (output)
			save_file(output_filename_robert_stream[i], pinned_filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f\n\n", speedup());

		hipFree(d_image);
		hipFree(d_filtered_image);
	}


	for (int i = 0; i < STREAMS; i++)
		hipStreamDestroy(stream[i]);

	free(image);
	hipHostFree(pinned_image);
	hipHostFree(pinned_filtered_image);
}

void stream_smem_robert_convolution_gpu(const char* filename, int kernel_side, int kernel_radius, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &filtered_image_size, &f_width, &f_height, kernel_radius);
	//Pinned memory allocation
	CHECK(hipHostAlloc(&pinned_image, image_size, 0));
	CHECK(hipHostAlloc(&pinned_filtered_image, filtered_image_size, 0));
	memcpy(pinned_image, image, image_size);

	//Chunk_size is the chunk of the input image wich is elaborated by the stream
	size_t chunk_size = (image_size / STREAMS);
	//Chunk_size_result is the chunk of data written by kernels in the output
	size_t chunk_size_result = filtered_image_size / STREAMS;

	//Stream creation
	hipStream_t stream[STREAMS];
	for (int i = 0; i < STREAMS; i++)
		CHECK(hipStreamCreate(&stream[i]));

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width + block.x - 1) / block.x, ((f_height / STREAMS) + block.y - 1) / block.y);

		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);
		printf("Streams: %d\n", STREAMS);
		int tile_side = block_sizes[i] + kernel_radius * 2;
		size_t tile_size = tile_side * tile_side;
		//Offset_input is the offset from which a kernel starts to read input image data
		int offset_input = 0;
		//Since the input potentially has more channels than the output(the output is always in grayscale), we need a different offset.
		int offset_output = 0;
		int row_offset;
		int image_offset;
		int filtered_image_offset;


		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_filtered_image, filtered_image_size));
		for (int j = 0; j < STREAMS; j++)
		{
			row_offset = j * (height / STREAMS);
			image_offset = j * width*(height / STREAMS)*channels;
			filtered_image_offset = j * (width - kernel_radius * 2)*(height / STREAMS);
			CHECK(hipMemcpyAsync(&d_image[offset_input], &pinned_image[offset_input], chunk_size, hipMemcpyHostToDevice, stream[j]));
			kernel_convolution_stream_smem << <grid, block, tile_size, stream[j] >> > (d_image, d_filtered_image, width, height, channels, tile_side, row_offset, image_offset, filtered_image_offset, kernel_side, kernel_radius, ROBERT_KERNEL_CODE_H);
			CHECK(hipMemcpyAsync(&pinned_filtered_image[offset_output], &d_filtered_image[offset_output], chunk_size_result, hipMemcpyDeviceToHost, stream[j]));
			offset_input += (int)(image_size) / STREAMS;
			offset_output += (int)chunk_size_result;
		}

		for (int j = 0; j < STREAMS; j++)
			CHECK(hipStreamSynchronize(stream[j]));
		end_timer();

		if (output)
			save_file(output_filename_robert_stream_smem[i], pinned_filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f\n\n", speedup());

		hipFree(d_image);
		hipFree(d_filtered_image);
	}

	for (int i = 0; i < STREAMS; i++)
		hipStreamDestroy(stream[i]);

	free(image);
	hipHostFree(pinned_image);
	hipHostFree(pinned_filtered_image);
}

void naive_module_gpu(const char * filename, int kernel_side, int kernel_radius, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &filtered_image_size, &f_width, &f_height, kernel_radius);
	filtered_image = (unsigned char*)malloc(filtered_image_size);

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width + block.x - 1) / block.x, (f_height + block.y - 1) / block.y);

		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);

		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_filtered_image, filtered_image_size));
		CHECK(hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice));
		kernel_module << < grid, block >> > (d_image, d_filtered_image, width, height, channels);
		CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(filtered_image, d_filtered_image, filtered_image_size, hipMemcpyDeviceToHost));

		end_timer();

		if (output)
			save_file(output_filename_module[i], filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f\n\n", speedup());

		hipFree(d_image);
		hipFree(d_filtered_image);
	}
	free(image);
	free(filtered_image);
}

void smem_module_gpu(const char * filename, int kernel_side, int kernel_radius, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &filtered_image_size, &f_width, &f_height, kernel_radius);
	filtered_image = (unsigned char*)malloc(filtered_image_size);

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width + block.x - 1) / block.x, (f_height + block.y - 1) / block.y);
		int tile_side = block_sizes[i] + kernel_radius * 2;
		size_t tile_size = tile_side * tile_side;

		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);

		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_filtered_image, filtered_image_size));
		CHECK(hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice));

		kernel_module_smem << < grid, block, tile_size >> > (d_image, d_filtered_image, width, height, channels, tile_side);

		CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(filtered_image, d_filtered_image, filtered_image_size, hipMemcpyDeviceToHost));

		end_timer();

		if (output)
			save_file(output_filename_module_smem[i], filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f\n\n", speedup());

		hipFree(d_image);
		hipFree(d_filtered_image);
	}
	free(image);
	free(filtered_image);
}

void stream_module_gpu(const char * filename, int kernel_side, int kernel_radius, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &filtered_image_size, &f_width, &f_height, kernel_radius);
	//Pinned memory allocation
	CHECK(hipHostAlloc(&pinned_image, image_size, 0));
	CHECK(hipHostAlloc(&pinned_filtered_image, filtered_image_size, 0));
	memcpy(pinned_image, image, image_size);

	//Chunk_size is the chunk of the input image wich is elaborated by the stream
	size_t chunk_size = image_size / STREAMS;
	//Chunk_size_result is the chunk of data written by kernels in the output
	size_t chunk_size_result = filtered_image_size / STREAMS;

	//Stream creation
	hipStream_t stream[STREAMS];
	for (int i = 0; i < STREAMS; i++)
		CHECK(hipStreamCreate(&stream[i]));

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width + block.x - 1) / block.x, ((f_height / STREAMS) + block.y - 1) / block.y);

		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);
		printf("Streams: %d\n", STREAMS);
		//Offset_input is the offset from which a kernel starts to read input image data
		int offset_input = 0;
		//Since the input potentially has more channels than the output(the output is always in grayscale), we need a different offset.
		int offset_output = 0;
		int row_offset;
		int image_offset;
		int filtered_image_offset;

		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_filtered_image, filtered_image_size));
		for (int j = 0; j < STREAMS; j++)
		{
			row_offset = j * (height / STREAMS);
			image_offset = j * width*(height / STREAMS)*channels;
			filtered_image_offset = j * (width - kernel_radius * 2)*(height / STREAMS);
			CHECK(hipMemcpyAsync(&d_image[offset_input], &pinned_image[offset_input], chunk_size, hipMemcpyHostToDevice, stream[j]));
			kernel_module_stream << <grid, block, 0, stream[j] >> > (d_image, d_filtered_image, width, height, channels, row_offset, image_offset, filtered_image_offset);
			CHECK(hipMemcpyAsync(&pinned_filtered_image[offset_output], &d_filtered_image[offset_output], chunk_size_result, hipMemcpyDeviceToHost, stream[j]));
			offset_input += (int)chunk_size;
			offset_output += (int)chunk_size_result;
		}

		for (int j = 0; j < STREAMS; j++)
			CHECK(hipStreamSynchronize(stream[j]));
		end_timer();

		if (output)
			save_file(output_filename_module_stream[i], pinned_filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f\n\n", speedup());

		hipFree(d_image);
		hipFree(d_filtered_image);
	}

	for (int i = 0; i < STREAMS; i++)
		hipStreamDestroy(stream[i]);

	free(image);
	hipHostFree(pinned_image);
	hipHostFree(pinned_filtered_image);
}

void stream_smem_module_gpu(const char * filename, int kernel_side, int kernel_radius, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &filtered_image_size, &f_width, &f_height, kernel_radius);
	//Pinned memory allocation
	CHECK(hipHostAlloc(&pinned_image, image_size, 0));
	CHECK(hipHostAlloc(&pinned_filtered_image, filtered_image_size, 0));
	memcpy(pinned_image, image, image_size);

	//Chunk_size is the chunk of the input image wich is elaborated by the stream
	size_t chunk_size = image_size / STREAMS;
	//Chunk_size_result is the chunk of data written by kernels in the output
	size_t chunk_size_result = filtered_image_size / STREAMS;

	//Stream creation
	hipStream_t stream[STREAMS];
	for (int i = 0; i < STREAMS; i++)
		CHECK(hipStreamCreate(&stream[i]));

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width + block.x - 1) / block.x, ((f_height / STREAMS) + block.y - 1) / block.y);
		int tile_side = block_sizes[i] + kernel_radius * 2;
		size_t tile_size = tile_side * tile_side;

		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block_sizes[i], block_sizes[i]);
		printf("Streams: %d\n", STREAMS);
		//Offset_input is the offset from which a kernel starts to read input image data
		int offset_input = 0;
		//Since the input potentially has more channels than the output(the output is always in grayscale), we need a different offset.
		int offset_output = 0;
		int row_offset;
		int image_offset;
		int filtered_image_offset;

		begin_timer();
		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_filtered_image, filtered_image_size));
		for (int j = 0; j < STREAMS; j++)
		{
			row_offset = j * (height / STREAMS);
			image_offset = j * width*(height / STREAMS)*channels;
			filtered_image_offset = j * (width - kernel_radius * 2)*(height / STREAMS);
			CHECK(hipMemcpyAsync(&d_image[offset_input], &pinned_image[offset_input], chunk_size, hipMemcpyHostToDevice, stream[j]));
			kernel_module_stream_smem << <grid, block, tile_size, stream[j] >> > (d_image, d_filtered_image, width, height, channels, tile_side, row_offset, image_offset, filtered_image_offset);
			CHECK(hipMemcpyAsync(&pinned_filtered_image[offset_output], &d_filtered_image[offset_output], chunk_size_result, hipMemcpyDeviceToHost, stream[j]));
			offset_input += (int)chunk_size;
			offset_output += (int)chunk_size_result;
		}

		for (int j = 0; j < STREAMS; j++)
			CHECK(hipStreamSynchronize(stream[j]));
		end_timer();

		if (output)
			save_file(output_filename_module_stream_smem[i], pinned_filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f\n\n", speedup());

		hipFree(d_image);
		hipFree(d_filtered_image);
	}

	for (int i = 0; i < STREAMS; i++)
		hipStreamDestroy(stream[i]);

	free(image);
	hipHostFree(pinned_image);
	hipHostFree(pinned_filtered_image);
}

void naive_canny_gpu(const char * filename, float sigma, int kernel_side, int kernel_radius, float low_threshold_ratio, float high_threshold_ratio, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &gaussian_image_size, &f_width_gaussian, &f_height_gaussian, kernel_radius);

	f_width = f_width_gaussian -2;
	f_height = f_height_gaussian - 2;
	filtered_image_size = f_width * f_height;

	size_t orientations_size = sizeof(float) * f_width*f_height;
	filtered_image = (unsigned char*)malloc(filtered_image_size);

	int strong_color = 255;
	int weak_color = 40;
	float high_threshold = high_threshold_ratio * strong_color;
	float low_threshold = low_threshold_ratio * high_threshold;

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width_gaussian + block.x - 1) / block.x, (f_height_gaussian + block.y - 1) / block.y);
		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block.x, block.y);
		begin_timer();
		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_gaussian_image, gaussian_image_size));
		CHECK(hipMalloc((void**)&d_module_image, filtered_image_size));
		CHECK(hipMalloc((void**)&d_non_max_image, filtered_image_size));
		CHECK(hipMalloc((void**)&d_orientations, orientations_size));
		CHECK(hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice));
		//Gaussian filter
		kernel_convolution << <grid, block >> > (d_image, d_gaussian_image, width, height, channels, kernel_side, kernel_radius, GAUSS_KERNEL_CODE);
		grid = dim3((f_width_gaussian + block.x - 1) / block.x, (f_width_gaussian + block.y - 1) / block.y);
		//Module and orientations
		kernel_module_orientation << <grid, block >> > (d_gaussian_image, d_module_image, d_orientations, f_width_gaussian, f_height_gaussian, 1);
		grid = dim3((f_width + block.x - 3) / block.x, (f_height + block.y - 3) / block.y);
		//Non max suppression
		kernel_non_max_suppression << <grid, block >> > (d_module_image, d_non_max_image, d_orientations, f_width, f_height, weak_color, strong_color, low_threshold, high_threshold);
		//Hysteresis
		kernel_hysteresis << <grid, block >> > (d_non_max_image, d_module_image, f_width, f_height, weak_color, strong_color);
		CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(filtered_image, d_module_image, filtered_image_size, hipMemcpyDeviceToHost));
		end_timer();

		if (output)
			save_file(output_filename_canny[i], filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f\n\n", speedup());

		CHECK(hipFree(d_image));
		CHECK(hipFree(d_gaussian_image));
		CHECK(hipFree(d_module_image));
		CHECK(hipFree(d_non_max_image));
		CHECK(hipFree(d_orientations));
	}

	free(image);
	free(filtered_image);
}

void smem_canny_gpu(const char * filename, float sigma, int kernel_side, int kernel_radius, float low_threshold_ratio, float high_threshold_ratio, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &gaussian_image_size, &f_width_gaussian, &f_height_gaussian, kernel_radius);

	f_width = f_width_gaussian - 2;
	f_height = f_height_gaussian - 2;

	filtered_image_size = f_width * f_height;

	orientations_size = sizeof(float) * f_width*f_height;
	filtered_image = (unsigned char*)malloc(filtered_image_size);

	int strong_color = 255;
	int weak_color = 40;
	float high_threshold = high_threshold_ratio * strong_color;
	float low_threshold = low_threshold_ratio * high_threshold;
	size_t tile_size;
	int tile_side;

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		dim3 grid = dim3((f_width_gaussian + block.x - 1) / block.x, (f_height_gaussian + block.y - 1) / block.y);
		tile_side = block_sizes[i] + kernel_radius * 2;
		tile_size = tile_side * tile_side;
		printf("Grid: %d, %d, %d\n", grid.x, grid.y, grid.z);
		printf("Blocks: %dx%d\n", block.x, block.y);
		begin_timer();
		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_gaussian_image, gaussian_image_size));
		CHECK(hipMalloc((void**)&d_module_image, filtered_image_size));
		CHECK(hipMalloc((void**)&d_non_max_image, filtered_image_size));
		CHECK(hipMalloc((void**)&d_orientations, orientations_size));
		CHECK(hipMemcpy(d_image, image, image_size, hipMemcpyHostToDevice));
		//Gaussian filter
		kernel_convolution_smem << <grid, block, tile_size >> > (d_image, d_gaussian_image, width, height, channels, tile_side, kernel_side, kernel_radius, GAUSS_KERNEL_CODE);
		//Module and orientations
		grid = dim3((f_width_gaussian + block.x - 1) / block.x, (f_width_gaussian + block.y - 1) / block.y);
		tile_side = block_sizes[i] + 2;
		tile_size = tile_side * tile_side;
		kernel_module_orientation_smem << <grid, block, tile_size >> > (d_gaussian_image, d_module_image, d_orientations, f_width_gaussian, f_height_gaussian, 1, tile_side);
		//Non max suppression
		kernel_non_max_suppression_smem << <grid, block, tile_size >> > (d_module_image, d_non_max_image, d_orientations, f_width, f_height, weak_color, strong_color, low_threshold, high_threshold, tile_side);
		//Hysteresis
		kernel_hysteresis_smem << <grid, block, tile_size >> > (d_non_max_image, d_module_image, f_width, f_height, weak_color, strong_color, tile_side);
		CHECK(hipDeviceSynchronize());
		CHECK(hipMemcpy(filtered_image, d_module_image, filtered_image_size, hipMemcpyDeviceToHost));
		end_timer();

		if (output)
			save_file(output_filename_canny_smem[i], filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f\n\n", speedup());

		CHECK(hipFree(d_image));
		CHECK(hipFree(d_gaussian_image));
		CHECK(hipFree(d_module_image));
		CHECK(hipFree(d_non_max_image));
		CHECK(hipFree(d_orientations));
	}

	free(image);
	free(filtered_image);
}

void stream_canny_gpu(const char * filename, float sigma, int kernel_side, int kernel_radius, float low_threshold_ratio, float high_threshold_ratio, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &gaussian_image_size, &f_width_gaussian, &f_height_gaussian, kernel_radius);

	f_width = f_width_gaussian -2;
	f_height = f_height_gaussian - 2;

	orientations_size = sizeof(float) * f_width*f_height;
	filtered_image_size = f_width * f_height;

	int strong_color = 255;
	int weak_color = 40;
	float high_threshold = high_threshold_ratio * strong_color;
	float low_threshold = low_threshold_ratio * high_threshold;

	//Pinned memory allocation
	CHECK(hipHostAlloc(&pinned_image, image_size, 0));
	CHECK(hipHostAlloc(&pinned_filtered_image, gaussian_image_size, 0));
	memcpy(pinned_image, image, image_size);

	//Chunk_size is the chunk of the input image wich is elaborated by the stream
	size_t chunk_size = image_size / STREAMS;
	//Chunk_size_result is the chunk of data written by kernels in the output
	size_t chunk_size_result = filtered_image_size / STREAMS;

	//Stream creation
	hipStream_t stream[STREAMS];
	for (int i = 0; i < STREAMS; i++)
		CHECK(hipStreamCreate(&stream[i]));

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		printf("Streams: %d\n", STREAMS);
		//Offset_input is the offset from which a kernel starts to read input image data
		int offset_input = 0;
		//Since the input potentially has more channels than the output(the output is always in grayscale), we need a different offset.
		int offset_output = 0;
		int row_offset;
		int image_offset;
		int filtered_image_offset;

		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_gaussian_image, gaussian_image_size));
		CHECK(hipMalloc((void**)&d_module_image, filtered_image_size));
		CHECK(hipMalloc((void**)&d_non_max_image, filtered_image_size));
		CHECK(hipMalloc((void**)&d_orientations, orientations_size));

		for (int j = 0; j < STREAMS; j++)
		{
			//Gaussian filtering
			dim3 grid = dim3((f_width_gaussian + block.x - 1) / block.x, ((f_height_gaussian / STREAMS) + block.y - 1) / block.y);
			row_offset = j * (height / STREAMS);
			image_offset = j * width*(height / STREAMS)*channels;
			filtered_image_offset = j * f_width_gaussian*(height / STREAMS);
			CHECK(hipMemcpyAsync(&d_image[offset_input], &pinned_image[offset_input], chunk_size, hipMemcpyHostToDevice, stream[j]));
			kernel_convolution_stream << <grid, block, 0, stream[j] >> > (d_image, d_gaussian_image, width, height, channels, row_offset, image_offset, filtered_image_offset, kernel_side, kernel_radius, GAUSS_KERNEL_CODE);
			//Module
			grid = dim3((f_width + block.x - 1) / block.x, ((f_height / STREAMS) + block.y - 1) / block.y);
			row_offset = j * (f_height_gaussian / STREAMS);
			image_offset = j * f_width_gaussian*(f_height_gaussian / STREAMS);
			filtered_image_offset = j * f_width*(f_height_gaussian / STREAMS);
			kernel_module_orientation_stream << <grid, block, 0, stream[j] >> > (d_gaussian_image, d_module_image, d_orientations, f_width_gaussian, f_height_gaussian, 1, row_offset, image_offset, filtered_image_offset);
			//Non max suppression
			row_offset = j * (f_height / STREAMS);
			image_offset = j * f_width*(f_height / STREAMS);
			kernel_non_max_suppression_stream << <grid, block, 0, stream[j] >> > (d_module_image, d_non_max_image, d_orientations, f_width, f_height, row_offset, image_offset, weak_color, strong_color, low_threshold, high_threshold);
			//Hysteresis
			kernel_hysteresis_stream << <grid, block, 0, stream[j] >> > (d_non_max_image, d_module_image, f_width, f_height, row_offset, image_offset, weak_color, strong_color);
			CHECK(hipMemcpyAsync(&pinned_filtered_image[offset_output], &d_module_image[offset_output], chunk_size_result, hipMemcpyDeviceToHost, stream[j]));
			offset_input += (int)chunk_size;
			offset_output += (int)chunk_size_result;
		}

		for (int j = 0; j < STREAMS; j++)
			CHECK(hipStreamSynchronize(stream[j]));
		end_timer();

		if (output)
			save_file(output_filename_canny_stream[i], pinned_filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f\n\n", speedup());

		hipFree(d_image);
		hipFree(d_module_image);
		hipFree(d_filtered_image);
		hipFree(d_non_max_image);
		hipFree(d_gaussian_image);
		hipFree(d_orientations);
	}


	for (int i = 0; i < STREAMS; i++)
		hipStreamDestroy(stream[i]);

	free(image);
	hipHostFree(pinned_image);
	hipHostFree(pinned_filtered_image);
}

void stream_smem_canny_gpu(const char * filename, float sigma, int kernel_side, int kernel_radius, float low_threshold_ratio, float high_threshold_ratio, bool output)
{
	image = load_file_details(filename, &width, &height, &channels, &image_size, &gaussian_image_size, &f_width_gaussian, &f_height_gaussian, kernel_radius);

	f_width = f_width_gaussian - 2;
	f_height = f_height_gaussian - 2;

	orientations_size = sizeof(float) * f_width*f_height;
	filtered_image_size = f_width * f_height;

	int strong_color = 255;
	int weak_color = 40;
	float high_threshold = high_threshold_ratio * strong_color;
	float low_threshold = low_threshold_ratio * high_threshold;

	//Pinned memory allocation
	CHECK(hipHostAlloc(&pinned_image, image_size, 0));
	CHECK(hipHostAlloc(&pinned_filtered_image, gaussian_image_size, 0));
	memcpy(pinned_image, image, image_size);

	//Chunk_size is the chunk of the input image wich is elaborated by the stream
	size_t chunk_size = image_size / STREAMS;
	//Chunk_size_result is the chunk of data written by kernels in the output
	size_t chunk_size_result = filtered_image_size / STREAMS;

	//Stream creation
	hipStream_t stream[STREAMS];
	for (int i = 0; i < STREAMS; i++)
		CHECK(hipStreamCreate(&stream[i]));

	for (int i = 0; i < BLOCK_SIZES; i++)
	{
		dim3 block = dim3(block_sizes[i], block_sizes[i]);
		printf("Streams: %d\n", STREAMS);
		//Offset_input is the offset from which a kernel starts to read input image data
		int offset_input = 0;
		//Since the input potentially has more channels than the output(the output is always in grayscale), we need a different offset.
		int offset_output = 0;
		int row_offset;
		int image_offset;
		int filtered_image_offset;

		begin_timer();

		CHECK(hipMalloc((void**)&d_image, image_size));
		CHECK(hipMalloc((void**)&d_gaussian_image, gaussian_image_size));
		CHECK(hipMalloc((void**)&d_module_image, filtered_image_size));
		CHECK(hipMalloc((void**)&d_non_max_image, filtered_image_size));
		CHECK(hipMalloc((void**)&d_orientations, orientations_size));

		for (int j = 0; j < STREAMS; j++)
		{
			//Gaussian filtering
			dim3 grid = dim3((f_width_gaussian + block.x - 1) / block.x, ((f_height_gaussian / STREAMS) + block.y - 1) / block.y);
			row_offset = j * (height / STREAMS);
			image_offset = j * width*(height / STREAMS)*channels;
			filtered_image_offset = j * f_width_gaussian*(height / STREAMS);
			CHECK(hipMemcpyAsync(&d_image[offset_input], &pinned_image[offset_input], chunk_size, hipMemcpyHostToDevice, stream[j]));
			kernel_convolution_stream << <grid, block, 0, stream[j] >> > (d_image, d_gaussian_image, width, height, channels, row_offset, image_offset, filtered_image_offset, kernel_side, kernel_radius, GAUSS_KERNEL_CODE);
			//Module
			grid = dim3((f_width + block.x - 1) / block.x, ((f_height / STREAMS) + block.y - 1) / block.y);
			row_offset = j * (f_height_gaussian / STREAMS);
			image_offset = j * f_width_gaussian*(f_height_gaussian / STREAMS);
			filtered_image_offset = j * f_width*(f_height_gaussian / STREAMS);
			kernel_module_orientation_stream << <grid, block, 0, stream[j] >> > (d_gaussian_image, d_module_image, d_orientations, f_width_gaussian, f_height_gaussian, 1, row_offset, image_offset, filtered_image_offset);
			//Non max suppression
			row_offset = j * (f_height / STREAMS);
			image_offset = j * f_width*(f_height / STREAMS);
			kernel_non_max_suppression_stream << <grid, block, 0, stream[j] >> > (d_module_image, d_non_max_image, d_orientations, f_width, f_height, row_offset, image_offset, weak_color, strong_color, low_threshold, high_threshold);
			//Hysteresis
			kernel_hysteresis_stream << <grid, block, 0, stream[j] >> > (d_non_max_image, d_module_image, f_width, f_height, row_offset, image_offset, weak_color, strong_color);
			CHECK(hipMemcpyAsync(&pinned_filtered_image[offset_output], &d_module_image[offset_output], chunk_size_result, hipMemcpyDeviceToHost, stream[j]));
			offset_input += (int)chunk_size;
			offset_output += (int)chunk_size_result;
		}

		for (int j = 0; j < STREAMS; j++)
			CHECK(hipStreamSynchronize(stream[j]));
		end_timer();

		if (output)
			save_file(output_filename_canny_stream[i], pinned_filtered_image, f_width, f_height, 1);
		printf("Time elapsed for memory allocation, computation and memcpy H2D and D2H:%f seconds\n", time_elapsed());
		printf("Speedup: %f\n\n", speedup());

		hipFree(d_image);
		hipFree(d_module_image);
		hipFree(d_filtered_image);
		hipFree(d_non_max_image);
		hipFree(d_gaussian_image);
		hipFree(d_orientations);
	}


	for (int i = 0; i < STREAMS; i++)
		hipStreamDestroy(stream[i]);

	free(image);
	hipHostFree(pinned_image);
	hipHostFree(pinned_filtered_image);
}
